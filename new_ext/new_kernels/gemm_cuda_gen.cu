#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "gemm_cuda.h"
#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAGuard.h>
#include <iostream>
#include <cstdio>


__global__ void quant_forward_mm(
	half* in_feats,
       	int* kernel,
       	half* scales,
       	int* zeros,
       	half* out_feats,
       	int in_channels,
       	int num_packed_channels,
	int num_in_feats
) {
    /*
      in_feats M, IC
      scales IC // 128, OC [float16]
      zeros  IC // 128, OC // 8 [int32]
      kernel IC, OC // 8 [int32]
      output IC, OC
      IC = IF
    */

    int order_map[] = {0, 2, 4, 6, 1, 3, 5, 7}; 

    // they have to be the same dimension for this to work
    const int blocksize = 8;

    const int groupsize = 128;

    // preload into shared memory which is 10x faster than load from GMEM
    __shared__ float s_weight[blocksize * blocksize * 8];
    __shared__ half s_feats[blocksize * blocksize];

    // need to make sure there is no repeated x-y pairs
    const int x = blockIdx.x * blocksize + threadIdx.x / blocksize;
    const int y = blockIdx.y * blocksize + threadIdx.x % blocksize;

    // row and column in block-space
    const int chunk_row = x / blocksize;
    const int chunk_column = y / blocksize;

    // position of the thread inside the moving block in either feats / kernel
    const int thread_row = threadIdx.x / blocksize;
    const int thread_column = threadIdx.x % blocksize;

    const int num_output_channels = num_packed_channels * 8;

    // all the blocks start on column 0
    half* in_feats_ptr = in_feats + chunk_row * in_channels * blocksize;

    // all the blocks start on row 0
    int* kernel_ptr = kernel + chunk_column * blocksize;

    if (x < num_in_feats && y < num_packed_channels) {
	// one column of kernel becomes 8 in the output due to dequantising
        float tmp_results[8] = {0.0};

	for (int shift = 0; shift < in_channels; shift+=blocksize) {
           // order not important for correctness but is important for coalescence
           s_feats[thread_row * blocksize + thread_column] = in_feats_ptr[thread_row * in_channels + thread_column];

	   int x_kernel_offset = shift + thread_row;
           int z_item = *(zeros + x_kernel_offset / groupsize * num_output_channels / 8 + y);
	   int w_item = kernel_ptr[thread_row * num_packed_channels + thread_column];

           // calculate and store the dequantized weights
	   for (int pos = 0; pos < 8; pos++) {
	       half s_item = *(scales + x_kernel_offset / groupsize * num_output_channels + y * 8 + order_map[pos]);

	       float zero = static_cast<float>((z_item >> 4 * pos) & 0xf);
	       float weight = static_cast<float>((w_item >> 4 * pos) & 0xf);

	       float scaled_zero = zero * __half2float(s_item);
	       float dequant = (weight * __half2float(s_item)) - scaled_zero;

	       // index chosen to remove bank conflicts (matrix as pos-row-column order)
               int idx = order_map[pos] * blocksize * blocksize + thread_row * 8 + thread_column;
	       s_weight[idx] = dequant;
           }

	   __syncthreads();

	   // advance the block forward
	   in_feats_ptr += blocksize;
	   kernel_ptr += blocksize * num_packed_channels;

	   // multiply the features and weights together
	   for (int blockpos=0; blockpos < blocksize; blockpos++) {
	       half f_item = s_feats[thread_row * blocksize + blockpos];

	       for (int pos=0; pos < 8; pos++) {
		   // block cause of high MIO throttling and instructions executed
		   float dequant = s_weight[pos * blocksize * blocksize + blockpos * 8 + thread_column];
	           float value = __half2float(f_item) * dequant;
	           tmp_results[pos] +=  value;
	       }
           }

	   // next loop will change the shared memory again
	   __syncthreads();
        }

	// write out the results for this position
        for (int pos=0; pos < 8; pos++) {
	    half* out_ptr = out_feats + x * num_output_channels + y * 8 + pos;
	    *(half*)(out_ptr) = __float2half(tmp_results[pos]);
        }
    }

}

// feats: M, IC
// kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// scaling_factors: IC // G, OC [float16]
// zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// out: M, OC
torch::Tensor gemm_forward_cuda_new(
    torch::Tensor _in_feats,
    torch::Tensor _kernel,
    torch::Tensor _scaling_factors,
    torch::Tensor _zeros)
{
    int num_in_feats = _in_feats.size(0);
    int num_in_channels = _in_feats.size(1);
    int num_out_channels = _kernel.size(1) * 8;
    int num_packed_channels = _kernel.size(1);

    const at::cuda::OptionalCUDAGuard device_guard(device_of(_in_feats));

    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());

    at::Tensor _out_feats = torch::zeros({num_in_feats, num_out_channels}, options);

    auto in_feats = reinterpret_cast<half*>(_in_feats.data_ptr<at::Half>());
    auto kernel = reinterpret_cast<int*>(_kernel.data_ptr<int>());
    auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());
    auto scaling_factors = reinterpret_cast<half*>(_scaling_factors.data_ptr<at::Half>());
    auto zeros = reinterpret_cast<int*>(_zeros.data_ptr<int>());

    if (_in_feats.size(1) != _kernel.size(0))
        throw std::invalid_argument("Kernel and input channels mismatch dim0");
    if (_scaling_factors.size(1) != _kernel.size(1) * 8)
        throw std::invalid_argument("Kernel and scaling factors mismatch dim1");

    if (num_in_channels != _scaling_factors.size(0) * 128)
        throw std::invalid_argument("Invalid scaling factors size (dim1)");
    if (num_out_channels != _scaling_factors.size(1))
        throw std::invalid_argument("Invalid scaling factors size (dim1)");

    if (_zeros.size(0) != _scaling_factors.size(0))
        throw std::invalid_argument("Invalid zeros size (dim0)");
    if (_zeros.size(1) * 8 != num_out_channels)
        throw std::invalid_argument("Invalid zeros size (dim1)");

    if (num_in_feats % 8 != 0) {
        throw std::invalid_argument("In feats must be divisible by 8");
    }

    if (num_packed_channels % 8 != 0) {
        throw std::invalid_argument("Packed channels must be divisible by 8");
    }

    int num_threads = 8 * 8;
    dim3 threads(num_threads);
    dim3 blocks((num_in_feats + 8 - 1) / 8, (num_packed_channels + 8 - 1) / 8);

    quant_forward_mm<<<blocks, threads>>>(in_feats, kernel, scaling_factors, zeros, out_feats, num_in_channels, num_packed_channels, num_in_feats);

    return _out_feats;
}

